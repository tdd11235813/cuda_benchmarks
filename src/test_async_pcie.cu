#include <vector>
#include <iostream>
#include "cuda_helper.cuh"

static constexpr int ITERATIONS = 10;

template<typename T>
void perform_test(int n) {
  hipStream_t stream0, stream1;
  CHECK_CUDA(hipStreamCreate(&stream0));
  CHECK_CUDA(hipStreamCreate(&stream1));

  size_t bytes = n*sizeof(T);
  std::cout << "n = " << n << " (" << 1.0*bytes/1048576 << " MB)\n";

  TimerCPU cpustart;
  double cpums;
  float gputime;
  hipEvent_t start, end;
  CHECK_CUDA(hipEventCreate(&start));
  CHECK_CUDA(hipEventCreate(&end));

  T* ddata;
  CHECK_CUDA(hipMalloc(&ddata, bytes));

  {
    // sync memcpy w vector.data()

    std::vector<T> data_in(n,1);
    std::vector<T> data_out(n,0);

    CHECK_CUDA(hipEventRecord(start));
    cpustart.startTimer();

    for(int r=0; r<ITERATIONS; r++) {
      CHECK_CUDA(hipMemcpy(ddata, data_in.data(), bytes, hipMemcpyHostToDevice));
      CHECK_CUDA(hipMemcpy(data_out.data(), ddata, bytes, hipMemcpyDeviceToHost));
    }

    cpums = cpustart.stopTimer();
    CHECK_CUDA(hipEventRecord(end));
    CHECK_CUDA(hipEventSynchronize(end));

    CHECK_CUDA( hipEventElapsedTime(&gputime, start, end) );
    std::cout << "Sync copy time on vector.data() = " << gputime/ITERATIONS << " ms (bandwidth = " << bytes/gputime*ITERATIONS*2e-6<<" GB/s)"
              << std::endl<<" [cpu call duration = "<<cpums/ITERATIONS<<" ms]"
              << std::endl;


    // hipMemcpyAsync w vector.data()

    CHECK_CUDA(hipEventRecord(start,stream0));
    cpustart.startTimer();

    for(int r=0; r<ITERATIONS; r++) {
      CHECK_CUDA(hipMemcpyAsync(ddata, data_in.data(), bytes, hipMemcpyHostToDevice, stream0));
      CHECK_CUDA(hipMemcpyAsync(data_out.data(), ddata, bytes, hipMemcpyDeviceToHost, stream1));
    }

    cpums = cpustart.stopTimer();
    CHECK_CUDA(hipEventRecord(end,stream1));
    CHECK_CUDA(hipEventSynchronize(end));

    CHECK_CUDA( hipEventElapsedTime(&gputime, start, end) );
    std::cout << "Async copy time on vector.data() = " << gputime/ITERATIONS << " ms (bandwidth = " << bytes/gputime*ITERATIONS*2e-6<<" GB/s)"
              <<std::endl<< " [cpu call duration = "<<cpums/ITERATIONS<<" ms]"
              << std::endl;
  }


  // hipMemcpyAsync w hipHostMalloc'd data
  T* data_pinned_in;
  T* data_pinned_out;
  CHECK_CUDA(hipHostMalloc(&data_pinned_in, bytes));
  CHECK_CUDA(hipHostMalloc(&data_pinned_out, bytes));

  CHECK_CUDA(hipDeviceSynchronize());
  CHECK_CUDA(hipEventRecord(start,stream0));
  cpustart.startTimer();

  for(int r=0; r<ITERATIONS; r++) {
    CHECK_CUDA(hipMemcpyAsync(ddata, data_pinned_in, bytes, hipMemcpyHostToDevice, stream0));
    CHECK_CUDA(hipMemcpyAsync(data_pinned_out, ddata, bytes, hipMemcpyDeviceToHost, stream1));
  }

  cpums = cpustart.stopTimer();
  CHECK_CUDA(hipEventRecord(end,stream1));
  CHECK_CUDA(hipEventSynchronize(end));

	CHECK_CUDA( hipEventElapsedTime(&gputime, start, end) );
  std::cout << "Async copy time (pinned) = " << gputime/ITERATIONS << " ms (bandwidth = " << bytes/gputime*ITERATIONS*2e-6<<" GB/s)"
            << std::endl<<" [cpu call duration = "<<cpums/ITERATIONS<<" ms]"
            << std::endl;

  CHECK_CUDA(hipFree(ddata));
  CHECK_CUDA(hipHostFree(data_pinned_in));
  CHECK_CUDA(hipHostFree(data_pinned_out));
	CHECK_CUDA( hipEventDestroy(start) );
	CHECK_CUDA( hipEventDestroy(end) );
	CHECK_CUDA( hipStreamDestroy(stream0) );
	CHECK_CUDA( hipStreamDestroy(stream1) );

}

int main(int argc, char** argv)
{

  int n = 1<<20;
  int dev = 0;
  if(argc>=2)
    n = atoi(argv[1]);
  if(argc==3)
    dev = atoi(argv[2]);

	CHECK_CUDA(hipSetDevice(dev));
  perform_test<int>(n);


	CHECK_CUDA(hipDeviceReset());
	return 0;
}
