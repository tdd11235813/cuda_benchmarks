#include "hip/hip_runtime.h"
#include "cuda_helper.cuh"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <cuda_device_runtime_api.h>
#include <hip/hip_cooperative_groups.h>
#include <iostream>
#include <stdexcept>
#include <limits>

using namespace cooperative_groups;

enum class MultiGrid {
  NO, YES
};

template<int TBlocksize, typename TGroup, typename T>
__device__
T reduce(TGroup group, T *x, int n) {

  __shared__ T sdata[TBlocksize];

  // obtain default "current thread block" group
  thread_block my_block = this_thread_block();

  int lane = my_block.thread_rank(); // index \in {0,blocksize-1}

  int i = blockIdx.x * TBlocksize + lane; // or: this_grid().thread_rank()

  sdata[lane] = x[i];
  i += group.size();

  // --------
  // Level 1: [multi] group reduce
  // --------

  // reduce per thread with increased ILP by 4x unrolling sum.
  // the thread of our block reduces its 4 grid-neighbored threads and advances by group-striding loop
  while (i+3*group.size() < n) {
    sdata[lane] += x[i] + x[i+group.size()] + x[i+2*group.size()] + x[i+3*group.size()];
    i += 4*group.size();
  }

  // doing the remaining blocks
  while(i<n) {
    sdata[lane] += x[i];
    i += group.size();
  }

  my_block.sync();

  // --------
  // Level 2: block + warp reduce
  // --------

#pragma unroll
  for(unsigned int bs=TBlocksize,
        bsup=(TBlocksize+1)/2; // ceil(TBlocksize/2.0)
      bs>1;
      bs=bs/2,
        bsup=(bs+1)/2) // ceil(bs/2.0)
  {
    bool cond = lane < bsup // only first half of block is working
               && (lane+bsup) < TBlocksize // index for second half must be in bounds
               && (this_grid().thread_rank()+bsup)<n; // if elem in second half has been initialized before
    if(cond)
    {
      sdata[lane] += sdata[lane + bsup];
    }
    my_block.sync();
  }
  return sdata[0];
}

template<int TBlocksize, typename T>
__global__
void kernel_reduce(T* x, T* y, int n)
{
  auto grid = this_grid();
  thread_block my_block = this_thread_block();

  if(grid.thread_rank()>=n)
    return;

  T block_result = reduce<TBlocksize>(grid, x, n);

  // store block result to gmem
  if (my_block.thread_rank() == 0)
    y[blockIdx.x] = block_result;
// or: y[my_block.group_index().x] = block_result;

  // grid synchronisation
  grid.sync();

  // --------
  // final reduce
  // - each block has written its result to gmem (data is coalesced)
  // - reduce the block results to the final value
  // - since we use coop kernels dynamic parallelism is not usable
  // --------

  // first block on first device
  if (blockIdx.x==0) {
    // reduce results of all the blocks stored in y
    T result = reduce<TBlocksize>(my_block, y, gridDim.x);
    // store result of reduction
    if(my_block.thread_rank() == 0)
      y[0] = result;
  }
}

template<int TBlocksize, typename T>
__global__
void kernel_reduce_multi(T* x, T* y, int n)
{
  auto grid = this_multi_grid(); // ! // cannot be used by hipLaunchCooperativeKernel (will not terminate)
  thread_block my_block = this_thread_block();

  if(grid.thread_rank()>=n)
    return;

  T block_result = reduce<TBlocksize>(grid, x, n);

  if (my_block.thread_rank() == 0)
    y[blockIdx.x + grid.grid_rank()*gridDim.x] = block_result; // !

  grid.sync();

  if (grid.grid_rank()==0 && blockIdx.x==0) { // !
    T result = reduce<TBlocksize>(my_block, y, grid.num_grids()*gridDim.x); // !
    if(my_block.thread_rank() == 0)
      y[0] = result;
  }
}

template<typename T, int TRuns, MultiGrid TMultiGrid>
void reduce(T init, size_t n, int dev) {

  CHECK_CUDA( hipSetDevice(dev) );
  hipDeviceProp_t prop;
  CHECK_CUDA( hipGetDeviceProperties(&prop, dev) );
  hipEvent_t cstart, cend;
  CHECK_CUDA(hipEventCreate(&cstart));
  CHECK_CUDA(hipEventCreate(&cend));
  hipStream_t cstream;
  CHECK_CUDA(hipStreamCreate(&cstream));


  std::cout << getCUDADeviceInformations(dev).str();
  if(!prop.cooperativeLaunch)
    throw std::runtime_error("Device must support cooperativeLaunch property.");
  else
    std::cout << ", \"cooperativeLaunch\", \"supported\"";

  if(TMultiGrid == MultiGrid::YES) {
    if(!prop.cooperativeMultiDeviceLaunch)
      throw std::runtime_error("Device must support cooperativeMultiDeviceLaunch property.");
    else
      std::cout << ", \"cooperativeMultiDeviceLaunch\", \"supported\"";
  }
  std::cout << std::endl;

  const int nr_dev = 1;

  dim3 threads( 128 );
  int nbsm=0;
  hipOccupancyMaxActiveBlocksPerMultiprocessor(&nbsm, kernel_reduce<128,T>, threads.x, 0);
  int numSMs;
  hipDeviceGetAttribute(&numSMs, hipDeviceAttributeMultiprocessorCount, dev);
  dim3 blocks( nbsm*numSMs ); // factor must not exceed max number of active blocks per SM, otherwise runtime error will occur
  if( blocks.x > (n-1)/threads.x+1 )
    blocks.x = (n-1)/threads.x+1;

  T* x;
  T* y;
  CHECK_CUDA( hipMallocManaged(&x, n*sizeof(T)) );
  CHECK_CUDA( hipMallocManaged(&y, nr_dev*blocks.x*sizeof(T)) );
  for (int i = 0; i < n; i++) {
    x[i] = init;
  }
  y[0] = 0;


  float milliseconds = 0;
  float min_ms = std::numeric_limits<float>::max();
  hipLaunchParams params[1];
  void* args[] = {(void*)&x, (void*)&y, (void*)&n};

  for(int r=0; r<TRuns; ++r) {
    CHECK_CUDA(hipEventRecord(cstart, cstream));

    // kernel<128><<<blocks, threads>>>(x, y, n);
    // NOTE: to use grid_groups we must use hipLaunchCooperativeKernel
    // NOTE: such kernels cannot make use of dynamic parallelism

    if( TMultiGrid == MultiGrid::NO ) {
      CHECK_CUDA( hipLaunchCooperativeKernel((const void*)(&kernel_reduce<128, T>),
                                              blocks, threads, args,
                                              (size_t)0/*smem*/,
                                              cstream) ); //(hipStream_t)0/*stream*/) );
    } else {
      params[0].func = (void*)(&kernel_reduce_multi<128, T>);
      params[0].gridDim = blocks;
      params[0].blockDim = threads;
      params[0].args = args;
      params[0].sharedMem = 0;
      params[0].stream = cstream; // cannot use the NULL stream
      CHECK_CUDA(hipLaunchCooperativeKernelMultiDevice(params, 1 /*numDevices*/));
    }
    CHECK_CUDA( hipEventRecord(cend, cstream) );
    CHECK_CUDA( hipEventSynchronize(cend) );
    CHECK_CUDA( hipEventElapsedTime(&milliseconds, cstart, cend) );
    if(milliseconds<min_ms)
      min_ms = milliseconds;
  }

  T result_gpu = y[0];

  std::cout << "Result (n = "<<n<<"):\n"
            << "GPU: " << result_gpu << " (min kernels time = "<< min_ms <<" ms)\n"
            << "expected: " << init*n <<"\n"
            << (init*n != result_gpu ? "MISMATCH!!" : "Success") << "\n"
            << "max bandwidth: "<<n*sizeof(T)/min_ms*1e-6<<" GB/s"
            << std::endl;


  CHECK_CUDA(hipFree(x));
  CHECK_CUDA(hipFree(y));
  CHECK_CUDA(hipEventDestroy(cstart));
  CHECK_CUDA(hipEventDestroy(cend));
  CHECK_CUDA(hipStreamDestroy(cstream));

}

int main(void)
{
  std::cout << "[Single-Grid]\n\n";
  reduce<int,5, MultiGrid::NO>(1, 1<<28, 0);
  std::cout << "\n[Multi-Grid]\n\n";
  reduce<int,5, MultiGrid::YES>(1, 1<<28, 0);
  CHECK_CUDA( hipDeviceReset() );
  return 0;
}
